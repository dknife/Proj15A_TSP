#include "hip/hip_runtime.h"

#include "kernel.cuh"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>


__host__ void callKernel(unsigned int size, int *c, const int *a, const int *b) {
	addKernel <<< 1, size >>> (c, a, b);
}

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

